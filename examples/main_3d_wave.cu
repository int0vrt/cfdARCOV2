#include "hip/hip_runtime.h"
/*
cfdARCO - high-level framework for solving systems of PDEs on multi-GPUs system
Copyright (C) 2024 cfdARCHO

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++, C#, and Java: https://pvs-studio.com

#include <iostream>
#include <chrono>
#include <thread>
#include <argparse/argparse.hpp>

#include "mesh3d.hpp"
#include "fvm3d.hpp"
#include "utils3d.hpp"

Eigen::Matrix<float, -1, 1> boundary_sine(Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const DT* dt_) {

    Eigen::Matrix<float, -1, 1> ret{arr};
    ret[mesh->square_node_coord_to_idx(mesh->_x * 0.1, mesh->_y * 0.1, mesh->_z * 0.1)] = std::sin(static_cast<float>(dt_->_current_time_step_int) * 0.2);
    ret[mesh->square_node_coord_to_idx(mesh->_x * 0.9, mesh->_y * 0.9, mesh->_z * 0.9)] = std::sin(static_cast<float>(dt_->_current_time_step_int) * 0.2);

    return ret;
}

__global__ void boundary_sine_k(float *a, size_t dt_itr, size_t n_1, size_t n_2) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == n_1 || idx == n_2) {
        a[idx] = std::sin(static_cast<float>(dt_itr) * 0.2);
    }
}

CudaDataMatrixD boundary_sine_cu(Mesh3D* mesh, CudaDataMatrixD& arr, const DT* dt_) {
    CudaDataMatrixD arr_n{arr};

    size_t n_1 = mesh->square_node_coord_to_idx(mesh->_x * 0.1, mesh->_y * 0.1, mesh->_z * 0.1);
    size_t n_2 = mesh->square_node_coord_to_idx(mesh->_x * 0.9, mesh->_y * 0.9, mesh->_z * 0.9);

    int blocksize = 1024;
    int nblocks = std::ceil(static_cast<float>(arr_n._size) / static_cast<float>(blocksize));
    boundary_sine_k<<<nblocks, blocksize>>>(arr_n.data.get(), dt_->_current_time_step_int, n_1, n_2);
    sync_device();

    return arr_n;
}


int main(int argc, char **argv) {
    SingleLibInitializer3D initializer{argc, argv};

    auto mesh = initializer.mesh;
    auto timesteps = initializer.timesteps;

    auto initial_zero = initial_with_val(mesh.get(), 0);
//    auto u = Variable(mesh.get(), initial_zero, boundary_sine, boundary_sine_cu, "u");
    auto u = Variable(mesh.get(), initial_zero, boundary_sine, "u");

    std::vector<Variable*> space_vars {&u};
    auto dt = DT(mesh.get(), UpdatePolicies::constant_dt, UpdatePolicies::constant_dt_cu, 0.1, space_vars);

    float c = 0.3;

    EquationTemplate equation_system = {
            {d2t(u), '=', c * c * (d2dx(u) + d2dy(u)), true},
    };

    std::vector<Variable*> all_vars {&u};
    auto equation = Equation(timesteps);
    initializer.init_store(all_vars);

    auto begin = std::chrono::steady_clock::now();
    equation.evaluate(all_vars, equation_system, &dt, initializer.visualize, all_vars);
    auto end = std::chrono::steady_clock::now();
    if (CFDArcoGlobalInit::get_rank() == 0) std::cout << std::endl << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[microseconds]" << std::endl;

    initializer.finalize();
    return 0;
}