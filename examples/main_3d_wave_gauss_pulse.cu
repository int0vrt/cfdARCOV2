#include "hip/hip_runtime.h"
/*
cfdARCO - high-level framework for solving systems of PDEs on multi-GPUs system
Copyright (C) 2024 cfdARCHO

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++, C#, and Java: https://pvs-studio.com

#include <iostream>
#include <chrono>
#include <thread>
#include <argparse/argparse.hpp>

#include "mesh3d.hpp"
#include "fvm3d.hpp"
#include "utils3d.hpp"

Eigen::Matrix<float, -1, 1> _boundary_copy_2d_via_3d(Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const Eigen::Matrix<float, -1, 1>& copy_var) {
    auto ret = Eigen::Matrix<float, -1, 1>{mesh->_num_nodes};
    int i = 0;

    float x_limit_lower = mesh->_dx + 0.001;
    float x_limit_upper = mesh->_lx - mesh->_dx - 0.001;
    float y_limit_lower = mesh->_dy + 0.001;
    float y_limit_upper = mesh->_ly - mesh->_dy - 0.001;

    for (auto& node : mesh->_nodes) {
        if (x_limit_lower < node->x() && node->x() < x_limit_upper && y_limit_lower < node->y() && node->y() < y_limit_upper) {
            ret(i) = arr(i);
        } else {
            ret(i) = copy_var(i);
        }
        ++i;
    }

    return ret;
}

//auto boundary_copy(const Eigen::Matrix<float, -1, 1>& copy_var) {
//    return [copy_var] (Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const DT* dt_) { return _boundary_copy_2d_via_3d(mesh, arr, copy_var); };
//}

Eigen::Matrix<float, -1, 1> _boundary_zero_2d_via_3d(Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr) {
    auto ret = Eigen::Matrix<float, -1, 1>{mesh->_num_nodes};
    int i = 0;

    int nvals = 1;

    float x_limit_lower = nvals * mesh->_dx + 0.001;
    float x_limit_upper = mesh->_lx - nvals * mesh->_dx - 0.001;
    float y_limit_lower = nvals * mesh->_dy + 0.001;
    float y_limit_upper = mesh->_ly - nvals * mesh->_dy - 0.001;

    for (auto& node : mesh->_nodes) {
        if (x_limit_lower < node->x() && node->x() < x_limit_upper && y_limit_lower < node->y() && node->y() < y_limit_upper) {
            ret(i) = arr(i);
        } else {
            ret(i) = 0;
        }
        ++i;
    }

    return ret;
}

auto boundary_zero(const Eigen::Matrix<float, -1, 1>& copy_var) {
    return [copy_var] (Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const DT* dt_) { return _boundary_zero_2d_via_3d(mesh, arr); };
}


//Eigen::Matrix<float, -1, 1> initial_u(Mesh3D* mesh) {
//    auto ret = Eigen::Matrix<float, -1, 1>{mesh->_num_nodes};
//    int i = 0;
//
//    float A = 1.0;
//    float sigma = 0.1;
//
//    for (int x = 0; x < mesh->_x; ++x) {
//        for (int y = 0; y < mesh->_y; ++y) {
//            float x_val = std::abs(0.5f - mesh->_lx / (float) mesh->_x * (float) x);
//            float y_val = std::abs(0.5f - mesh->_ly / (float) mesh->_y * (float) y);
////            float x_val = (mesh->_lx / (float) mesh->_x * (float) x) - 0.5f;
////            float y_val = (mesh->_ly / (float) mesh->_y * (float) y) - 0.5f;
//            float res = A * std::exp(-(x_val * x_val + y_val * y_val) / (2 * sigma * sigma));
//            ret(i) = res;
//
//            i++;
//        }
//    }
//
//    return ret;
//}

#include <cmath>

Eigen::Matrix<float, -1, 1> initial_u(Mesh3D* mesh) {
    auto ret = Eigen::Matrix<float, -1, 1>{mesh->_num_nodes};
    int i = 0;

    float A = 1.0;
    float sigma = 0.1;

    for (int x = 0; x < mesh->_x; ++x) {
        for (int y = 0; y < mesh->_y; ++y) {
//            float x_val = std::abs(0.5f - mesh->_lx / (float) mesh->_x * (float) x);
//            float y_val = std::abs(0.5f - mesh->_ly / (float) mesh->_y * (float) y);
            float y_val = mesh->_ly / (float) mesh->_y * (float) y;
            float x_val = mesh->_lx / (float) mesh->_x * (float) x;
            float res = A * std::sin(M_PI * x_val) * std::sin(M_PI * y_val);
            ret(i) = res;

            i++;
        }
    }

    return ret;
}

Eigen::Matrix<float, -1, 1> _boundary_copy(Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const Eigen::Matrix<float, -1, 1>& copy_var) {
    auto arr1 = arr.cwiseProduct(mesh->_node_is_boundary_reverse);
    auto copy_var1 = copy_var.cwiseProduct(mesh->_node_is_boundary);
    return arr1 + copy_var1;
}

inline auto boundary_copy(const Eigen::Matrix<float, -1, 1>& copy_var) {
    return [copy_var] (Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const DT* dt_) { return _boundary_copy(mesh, arr, copy_var); };
}

CudaDataMatrixD _boundary_copy_cu(Mesh3D* mesh_in, CudaDataMatrixD& arr, const CudaDataMatrixD& copy_var, const DT* dt_) {
    auto* mesh = dynamic_cast<CudaMesh3D*>(mesh_in);
    auto arr1 = arr * mesh->_node_is_boundary_reverse_cu;
    auto copy_var1 = copy_var * mesh->_node_is_boundary_cu;
    return arr1 + copy_var1;
}

inline auto boundary_copy_cu(const Eigen::Matrix<float, -1, 1>& copy_var) {
    CudaDataMatrixD cuda_copy_var;
    if (CFDArcoGlobalInit::cuda_enabled) {
        cuda_copy_var = CudaDataMatrixD::from_eigen(copy_var);
    }

    return [cuda_copy_var](Mesh3D *mesh, CudaDataMatrixD &arr, const DT* dt_) {
        if (CFDArcoGlobalInit::cuda_enabled)
            return _boundary_copy_cu(mesh, arr, cuda_copy_var, dt_);
        else
            return cuda_copy_var;
    };
}

Eigen::Matrix<float, -1, 1> boundary_sine(Mesh3D* mesh, Eigen::Matrix<float, -1, 1>& arr, const DT* dt_) {

    Eigen::Matrix<float, -1, 1> ret{arr};
//    ret[mesh->square_node_coord_to_idx(mesh->_x * 0.1, mesh->_y * 0.1, mesh->_z * 0.1)] = std::sin(static_cast<float>(dt_->_current_time_step_int) * 0.);
//    ret[mesh->square_node_coord_to_idx(mesh->_x * 0.9, mesh->_y * 0.9, mesh->_z * 0.9)] = std::sin(static_cast<float>(dt_->_current_time_step_int) * 0.);

    return ret;
}

int main(int argc, char **argv) {
    SingleLibInitializer3D initializer{argc, argv};

    auto mesh = initializer.mesh;
    auto timesteps = initializer.timesteps;

    auto initial_ = initial_u(mesh.get());
    initial_ = _boundary_zero_2d_via_3d(mesh.get(), initial_);
    auto _zero = initial_with_val(mesh.get(), 0);
//    auto u = Variable(mesh.get(), initial_, boundary_copy(_zero), boundary_copy_cu(_zero), "u");
    auto u = Variable(mesh.get(), initial_, boundary_zero(_zero), "u");

    std::vector<Variable*> space_vars {&u};
    auto dt = DT(mesh.get(), UpdatePolicies::constant_dt, UpdatePolicies::constant_dt_cu, initializer.dt, space_vars);

    float c = 0.3;

    EquationTemplate equation_system = {
            {d2t(u), '=', c * c * (d2dx(u) + d2dy(u)), false},
    };

    std::vector<Variable*> all_vars {&u};
    auto equation = Equation(timesteps);
    initializer.init_store(all_vars);

    auto begin = std::chrono::steady_clock::now();
    equation.evaluate(all_vars, equation_system, &dt, initializer.visualize, all_vars);
    auto end = std::chrono::steady_clock::now();
    if (CFDArcoGlobalInit::get_rank() == 0) std::cout << std::endl << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[microseconds]" << std::endl;

    initializer.finalize();
    return 0;
}