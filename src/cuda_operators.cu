#include "hip/hip_runtime.h"
/*
cfdARCO - high-level framework for solving systems of PDEs on multi-GPUs system
Copyright (C) 2024 cfdARCHO

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "cuda_operators.hpp"
#include "decls.hpp"
#include <thrust/pair.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <custom_cuda_functions.hpp>

#define BLOCK_SIZE 64

__global__ void add_mtrx_k(const float *a, const float *b, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

CudaDataMatrixD add_mtrx(const CudaDataMatrixD &a, const CudaDataMatrixD &b) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    add_mtrx_k<<<nblocks, blocksize>>>(a.data.get(), b.data.get(), res.data.get(), a._size);

    return res;
}

__global__ void sub_mtrx_k(const float *a, const float *b, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] - b[idx];
    }
}

CudaDataMatrixD sub_mtrx(const CudaDataMatrixD &a, const CudaDataMatrixD &b) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    sub_mtrx_k<<<nblocks, blocksize>>>(a.data.get(), b.data.get(), res.data.get(), a._size);

    return res;
}

__global__ void mul_mtrx_k(const float *a, const float *b, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

CudaDataMatrixD mul_mtrx(const CudaDataMatrixD &a, const CudaDataMatrixD &b) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    mul_mtrx_k<<<nblocks, blocksize>>>(a.data.get(), b.data.get(), res.data.get(), a._size);

    return res;
}

__global__ void mul_mtrx_by_float_k(const float *a, float b, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b;
    }
}

CudaDataMatrixD mul_mtrx(const CudaDataMatrixD &a, const float b) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    mul_mtrx_by_float_k<<<nblocks, blocksize>>>(a.data.get(), b, res.data.get(), a._size);
    return res;
}

__global__ void div_mtrx_k(const float *a, const float *b, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] / b[idx];
    }
}

CudaDataMatrixD div_mtrx(const CudaDataMatrixD &a, const CudaDataMatrixD &b) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    div_mtrx_k<<<nblocks, blocksize>>>(a.data.get(), b.data.get(), res.data.get(), a._size);

    return res;
}

__global__ void div_const_k(const float *a, const float b, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] / b;
    }
}

CudaDataMatrixD div_const(const CudaDataMatrixD &a, const float b) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    div_const_k<<<nblocks, blocksize>>>(a.data.get(), b, res.data.get(), a._size);

    return res;
}

__global__ void neg_mtrx_k(const float *a, float *c, int n) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = -a[idx];
    }
}

CudaDataMatrixD neg_mtrx(const CudaDataMatrixD &a) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};
    neg_mtrx_k<<<nblocks, blocksize>>>(a.data.get(), res.data.get(), a._size);

    return res;
}

__global__ void cfl_cu_k(
        float dl, float gamma,
        const float * __restrict__ p_in,
        const float * __restrict__ rho_in,
        const float * __restrict__ u_in,
        const float * __restrict__ v_in,
        const float * __restrict__ w_in,
        int rows,
        float * __restrict__ value_memory
) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < rows) {
        auto p = p_in[idx];
        auto rho = rho_in[idx];
        auto u = u_in[idx];
        auto v = v_in[idx];
        auto w = w_in[idx];

        auto p1 = sqrt((gamma * p) / rho);
        auto p2 = sqrt(u * u + v * v + w * w);
        auto result = dl * (1.0 / (p1 + p2));
        value_memory[idx] = result;
    }
}

float cfl_cu(float dl, float gamma, const CudaDataMatrixD &p, const CudaDataMatrixD &rho, const CudaDataMatrixD &u,
              const CudaDataMatrixD &v, const CudaDataMatrixD &w) {
    int rows = p._size;
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(rows) / static_cast<float>(blocksize));

    CudaDataMatrixD memory_value{static_cast<size_t>(rows)};

    cfl_cu_k<<<nblocks, blocksize>>>(
            dl,
            gamma,
            p.data.get(),
            rho.data.get(),
            u.data.get(),
            v.data.get(),
            w.data.get(),
            rows,
            memory_value.data.get()
    );

    sync_device();
    thrust::device_ptr<float> ptr = thrust::device_pointer_cast<float>(memory_value.data.get());
    auto min_ptr = thrust::min_element(ptr, ptr + rows);

    return *min_ptr;
}

__global__ void cfl_cu_k(
        float dl, float gamma,
        const float * __restrict__ p_in,
        const float * __restrict__ rho_in,
        const float * __restrict__ u_in,
        int rows,
        float * __restrict__ value_memory
) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < rows) {
        auto p = p_in[idx];
        auto rho = rho_in[idx];
        auto u = u_in[idx];

        auto p1 = sqrt((gamma * p) / rho);
        auto p2 = sqrt(u * u);
        auto result = dl * (1.0 / (p1 + p2));
        value_memory[idx] = result;
    }
}

float cfl_cu(float dl, float gamma, const CudaDataMatrixD &p, const CudaDataMatrixD &rho, const CudaDataMatrixD &u) {
    int rows = p._size;
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(rows) / static_cast<float>(blocksize));

    CudaDataMatrixD memory_value{static_cast<size_t>(rows)};

    cfl_cu_k<<<nblocks, blocksize>>>(
            dl,
            gamma,
            p.data.get(),
            rho.data.get(),
            u.data.get(),
            rows,
            memory_value.data.get()
    );

    sync_device();
    thrust::device_ptr<float> ptr = thrust::device_pointer_cast<float>(memory_value.data.get());
    auto min_ptr = thrust::min_element(ptr, ptr + rows);

    return *min_ptr;
}

template<typename MeshClass>
__global__ void eval_grad_k(const float * __restrict__ var_ptr,
                          const size_t * __restrict__ ids_ptr,
                          const float * __restrict__ normals_x_ptr,
                          const float * __restrict__ normals_y_ptr,
                          const float * __restrict__ normals_z_ptr,
                          const float * __restrict__ face_area_ptr,
                          const float * __restrict__ volume_ptr,
                          float * __restrict__ ret_ptr,
                          bool clc_x, bool clc_y, bool clc_z,
                          size_t n) {

    float interpolation_ret[MeshClass::n_faces];
    size_t face_neigh_ids[MeshClass::n_faces];
    float face_area[MeshClass::n_faces];
    float var_neigh[MeshClass::n_faces];
    float normals_face[MeshClass::n_dims][MeshClass::n_faces];
    const float* normals_ptr[MeshClass::n_dims] = {
            normals_x_ptr,
            normals_y_ptr,
            normals_z_ptr
    };
    float grad[MeshClass::n_dims];

    auto idx = (size_t) blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float crr = read_scalar_from_self(var_ptr, idx, n);
        float volume = read_scalar_from_self(volume_ptr, idx, n);

        read_n_vars_from_self<MeshClass::n_faces>(ids_ptr, face_neigh_ids, idx, n);
        read_normals_from_self<MeshClass>(normals_ptr, normals_face, idx, n);
        read_n_vars_from_self<MeshClass::n_faces>(face_area_ptr, face_area, idx, n);

        read_n_vars_from_neigh<MeshClass>(var_ptr, face_neigh_ids, var_neigh);

        interpolate_to_face_linear_cu_k<MeshClass>(crr, var_neigh, interpolation_ret);
        gauss_grad_cu_k<MeshClass>(interpolation_ret, normals_face, face_area, volume, grad);

        float accum[1];
        accum[0] = 0.0;
        if (clc_x) accum[0] += grad[0];
        if (clc_y) accum[0] += grad[1];
        if (clc_z) accum[0] += grad[2];

        write_n_vars_to_self<1>(ret_ptr, accum, idx, n);
    }
}

CudaDataMatrixD eval_grad(CudaMesh3D* mesh, const CudaDataMatrixD &a, bool clc_x, bool clc_y, bool clc_z) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};

    eval_grad_k<Mesh3D><<<nblocks, blocksize>>>(
            a.data.get(),
            mesh->_ids_cu.data.get(),

            mesh->_normal_alt_x_cu.data.get(),
            mesh->_normal_alt_y_cu.data.get(),
            mesh->_normal_alt_z_cu.data.get(),

            mesh->_face_areas_cu.data.get(),
            mesh->_volumes_cu.data.get(),
            res.data.get(),
            clc_x, clc_y, clc_z,
            a._size);
    sync_device();
    return res;
}

template<typename MeshClass>
__global__ void eval_stab_first_stage_k(const float * __restrict__ var_ptr,
                            const size_t * __restrict__ ids_ptr,
                            const float * __restrict__ normals_x_ptr,
                            const float * __restrict__ normals_y_ptr,
                            const float * __restrict__ normals_z_ptr,
                            const float * __restrict__ face_area_ptr,
                            const float * __restrict__ volume_ptr,
                            const float * __restrict__ len_node_center_to_face_ptr,
                            float * __restrict__ ret_x_ptr,
                            float * __restrict__ ret_y_ptr,
                            float * __restrict__ ret_z_ptr,
                            bool clc_x, bool clc_y, bool clc_z,
                            size_t n) {

    float interpolation_ret[MeshClass::n_faces];
    float len_node_center_to_face[MeshClass::n_faces];
    size_t face_neigh_ids[MeshClass::n_faces];
    float face_area[MeshClass::n_faces];
    float var_neigh[MeshClass::n_faces];
    float normals_face[MeshClass::n_dims][MeshClass::n_faces];
    const float* normals_ptr[MeshClass::n_dims] = {
            normals_x_ptr,
            normals_y_ptr,
            normals_z_ptr
    };
    float grad[MeshClass::n_dims];
    float interpolated_upwing[MeshClass::n_dims][MeshClass::n_faces];

    auto idx = (size_t) blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float crr = read_scalar_from_self(var_ptr, idx, n);
        float volume = read_scalar_from_self(volume_ptr, idx, n);

        read_n_vars_from_self<MeshClass::n_faces>(len_node_center_to_face_ptr, len_node_center_to_face, idx, n);
        read_n_vars_from_self<MeshClass::n_faces>(ids_ptr, face_neigh_ids, idx, n);
        read_normals_from_self<MeshClass>(normals_ptr, normals_face, idx, n);
        read_n_vars_from_self<MeshClass::n_faces>(face_area_ptr, face_area, idx, n);

        read_n_vars_from_neigh<MeshClass>(var_ptr, face_neigh_ids, var_neigh);

        interpolate_to_face_linear_cu_k<MeshClass>(crr, var_neigh, interpolation_ret);
        gauss_grad_cu_k<MeshClass>(interpolation_ret, normals_face, face_area, volume, grad);

        interpolate_to_face_upwing_cu_k<MeshClass>(crr, grad, len_node_center_to_face, normals_face, interpolated_upwing);

        write_n_vars_to_self<MeshClass::n_faces>(ret_x_ptr, interpolated_upwing[0], idx, n);
        write_n_vars_to_self<MeshClass::n_faces>(ret_y_ptr, interpolated_upwing[1], idx, n);
        write_n_vars_to_self<MeshClass::n_faces>(ret_z_ptr, interpolated_upwing[2], idx, n);
    }
}

template<typename MeshClass>
__global__ void eval_stab_second_stage_k(
                                        const float * __restrict__ interpolated_upwing_x_prt,
                                        const float * __restrict__ interpolated_upwing_y_prt,
                                        const float * __restrict__ interpolated_upwing_z_prt,
                                        const size_t * __restrict__ ids_ptr,
                                        const float * __restrict__ normals_x_ptr,
                                        const float * __restrict__ normals_y_ptr,
                                        const float * __restrict__ normals_z_ptr,
                                        const float * __restrict__ face_area_ptr,
                                        const float * __restrict__ volume_ptr,
                                        const float * __restrict__ len_node_center_to_face_ptr,
                                        float * __restrict__ ret_ptr,
                                        bool clc_x, bool clc_y, bool clc_z,
                                        size_t n) {

    float interpolated_upwing[MeshClass::n_dims][MeshClass::n_faces];
    float partial_res[MeshClass::n_dims];
    float summed_faces[MeshClass::n_faces];
    float interpolated_upwing_collected[MeshClass::n_dims][MeshClass::n_faces];
    float len_node_center_to_face[MeshClass::n_faces];
    size_t face_neigh_ids[MeshClass::n_faces];
    float face_area[MeshClass::n_faces];
    float var_neigh[MeshClass::n_faces];
    float normals_face[MeshClass::n_dims][MeshClass::n_faces];
    const float* normals_ptr[MeshClass::n_dims] = {
            normals_x_ptr,
            normals_y_ptr,
            normals_z_ptr
    };
    const float* interpolated_upwing_ptr[MeshClass::n_dims] = {
            interpolated_upwing_x_prt,
            interpolated_upwing_y_prt,
            interpolated_upwing_z_prt
    };
    const bool dm_enabled[] = {
            clc_x,
            clc_y,
            clc_z
    };
    float grad[MeshClass::n_dims];

    float accum[1];
    accum[0] = 0.0;

    auto idx = (size_t) blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float volume = read_scalar_from_self(volume_ptr, idx, n);

        read_n_vars_from_self<MeshClass::n_faces>(ids_ptr, face_neigh_ids, idx, n);
        read_n_vars_from_self<MeshClass::n_faces>(len_node_center_to_face_ptr, len_node_center_to_face, idx, n);
#pragma unroll
        for (int dm = 0; dm < MeshClass::n_dims; ++dm) {
            read_n_vars_from_self<MeshClass::n_faces>(normals_ptr[dm], normals_face[dm], idx, n);
            read_n_vars_from_self<MeshClass::n_faces>(interpolated_upwing_ptr[dm], interpolated_upwing[dm], idx, n);
        }
        read_n_vars_from_self<MeshClass::n_faces>(face_area_ptr, face_area, idx, n);

        read_face_vars_from_neigh_opposite_face<MeshClass>(interpolated_upwing_ptr, face_neigh_ids, interpolated_upwing_collected, n);


#pragma unroll
        for (int dm = 0; dm < MeshClass::n_dims; ++dm) {
            if (!dm_enabled[dm]) continue;

#pragma unroll
            for (int fc = 0; fc < MeshClass::n_faces; ++fc) {
                summed_faces[fc] = 0.5 * (interpolated_upwing[dm][fc] - interpolated_upwing_collected[dm][fc]);
            }
            gauss_grad_cu_k<MeshClass>(summed_faces, normals_face, face_area, volume, partial_res);
            accum[0] += partial_res[dm];
        }

        write_n_vars_to_self<1>(ret_ptr, accum, idx, n);
    }
}

CudaDataMatrixD eval_stab(CudaMesh3D* mesh, const CudaDataMatrixD &a, bool clc_x, bool clc_y, bool clc_z) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};

    CudaDataMatrixD interpolated_upwing_x{a._size * CudaMesh3D::n_faces};
    CudaDataMatrixD interpolated_upwing_y{a._size * CudaMesh3D::n_faces};
    CudaDataMatrixD interpolated_upwing_z{a._size * CudaMesh3D::n_faces};

    eval_stab_first_stage_k<Mesh3D><<<nblocks, blocksize>>>(
            a.data.get(),
            mesh->_ids_cu.data.get(),

            mesh->_normal_alt_x_cu.data.get(),
            mesh->_normal_alt_y_cu.data.get(),
            mesh->_normal_alt_z_cu.data.get(),

            mesh->_face_areas_cu.data.get(),
            mesh->_volumes_cu.data.get(),
            mesh->_len_node_center_to_face_cu.data.get(),

            interpolated_upwing_x.data.get(),
            interpolated_upwing_y.data.get(),
            interpolated_upwing_z.data.get(),

            clc_x, clc_y, clc_z,
            a._size);
    sync_device();

    eval_stab_second_stage_k<Mesh3D><<<nblocks, blocksize>>>(
            interpolated_upwing_x.data.get(),
            interpolated_upwing_y.data.get(),
            interpolated_upwing_z.data.get(),

            mesh->_ids_cu.data.get(),

            mesh->_normal_x_cu.data.get(),
            mesh->_normal_y_cu.data.get(),
            mesh->_normal_z_cu.data.get(),

            mesh->_face_areas_cu.data.get(),
            mesh->_volumes_cu.data.get(),
            mesh->_len_node_center_to_face_cu.data.get(),

            res.data.get(),

            clc_x, clc_y, clc_z,
            a._size);
    sync_device();

    return res;
}


template<typename MeshClass>
__global__ void eval_grad2_k(const float * __restrict__ var_ptr,
                          const size_t * __restrict__ ids_ptr,
                          const float * __restrict__ normals_alt_x_ptr,
                          const float * __restrict__ normals_alt_y_ptr,
                          const float * __restrict__ normals_alt_z_ptr,
                         const float * __restrict__ normals_x_ptr,
                         const float * __restrict__ normals_y_ptr,
                         const float * __restrict__ normals_z_ptr,
                          const float * __restrict__ face_area_ptr,
                          const float * __restrict__ alpha_d_ptr,
                          const float * __restrict__ volume_ptr,
                          float * __restrict__ ret_ptr,
                          bool clc_x, bool clc_y, bool clc_z,
                          size_t n) {

    size_t face_neigh_ids[MeshClass::n_faces];
    float face_area[MeshClass::n_faces];
    float alpha_d[MeshClass::n_faces];
    float var_neigh[MeshClass::n_faces];
    float normals_face[MeshClass::n_dims][MeshClass::n_faces];
    const float* normals_ptr[MeshClass::n_dims] = {
            normals_x_ptr,
            normals_y_ptr,
            normals_z_ptr
    };
    float corrected_surface_normal_grad[MeshClass::n_faces];
    float lapl[MeshClass::n_dims];

    auto idx = (size_t) blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float crr = read_scalar_from_self(var_ptr, idx, n);
        float volume = read_scalar_from_self(volume_ptr, idx, n);

        read_n_vars_from_self<MeshClass::n_faces>(ids_ptr, face_neigh_ids, idx, n);
        read_n_vars_from_self<MeshClass::n_faces>(face_area_ptr, face_area, idx, n);
        read_n_vars_from_self<MeshClass::n_faces>(alpha_d_ptr, alpha_d, idx, n);

        for (int dm = 0; dm < MeshClass::n_dims; ++dm) {
            read_n_vars_from_self<MeshClass::n_faces>(normals_ptr[dm], normals_face[dm], idx, n);
        }

        read_n_vars_from_neigh<MeshClass>(var_ptr, face_neigh_ids, var_neigh);
        corrected_surface_normal_grad_cu_k<MeshClass>(crr, var_neigh, alpha_d, corrected_surface_normal_grad);
        gauss_grad_cu_k<MeshClass>(corrected_surface_normal_grad, normals_face, face_area, volume, lapl);

        float accum[1];
        accum[0] = 0.0;
        if (clc_x) accum[0] += lapl[0];
        if (clc_y) accum[0] += lapl[1];
        if (clc_z) accum[0] += lapl[2];

        write_n_vars_to_self<1>(ret_ptr, accum, idx, n);
    }
}

CudaDataMatrixD eval_grad2(CudaMesh3D* mesh, const CudaDataMatrixD &a, bool clc_x, bool clc_y, bool clc_z) {
    int blocksize = BLOCK_SIZE;
    int nblocks = std::ceil(static_cast<float>(a._size) / static_cast<float>(blocksize));
    CudaDataMatrixD res{a._size};

    eval_grad2_k<Mesh3D><<<nblocks, blocksize>>>(a.data.get(),
                                       mesh->_ids_cu.data.get(),

                                        mesh->_normal_alt_x_cu.data.get(),
                                        mesh->_normal_alt_y_cu.data.get(),
                                        mesh->_normal_alt_z_cu.data.get(),
                                        mesh->_normal_x_cu.data.get(),
                                        mesh->_normal_y_cu.data.get(),
                                        mesh->_normal_z_cu.data.get(),

                                       mesh->_face_areas_cu.data.get(),
                                       mesh->_alpha_d_cu.data.get(),
                                       mesh->_volumes_cu.data.get(),
                                       res.data.get(),
                                       clc_x, clc_y, clc_z,
                                       a._size);
    sync_device();
    return res;
}

